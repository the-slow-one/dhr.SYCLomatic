#include "hip/hip_runtime.h"
// RUN: mkdir -p %T/common
// RUN: cd %T
// RUN: cat %S/compile_commands.json > %T/compile_commands.json
// RUN: cat %S/common/test1.cpp > %T/common/test1.cpp
// RUN: cat %S/test2.cpp > %T/test2.cpp
// RUN: cat %S/test3.cu > %T/test3.cu
// RUN: cat %S/test.h > %T/test.h
// RUN: dpct --in-root=%T --out-root=%T/out -p %T --format-range=none --cuda-include-path="%cuda-path/include" --enable-codepin
// RUN: FileCheck %S/common/test1.cpp --match-full-lines --input-file %T/out_codepin_cuda/common/test1.cpp
#include "test.h"

__global__ void kernel(){
    float a = float_to_force;
}

int main() {
    kernel<<<1, 1>>>();
    return 0;      
}