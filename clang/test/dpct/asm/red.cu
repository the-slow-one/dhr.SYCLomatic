// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/red %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/red/red.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/red/red.dp.cpp -o %T/red/red.dp.o %}

// clang-format off
#include <cstdint>
#include <hip/hip_runtime.h>

// CHECK: void atomicAddKernel(int* lock, int val, const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::plus<>());
// CHECK-NEXT:}
__global__ void atomicAddKernel(int* lock, int val) {
    asm volatile("red.relaxed.gpu.global.add.s32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicOrKernel(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                     const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_or<>());
// CHECK-NEXT:}
__global__ void atomicOrKernel(uint32_t* lock, uint32_t val) {
    asm volatile("red.relaxed.gpu.global.or.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicXorKernel(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_xor<>());
// CHECK-NEXT:}
__global__ void atomicXorKernel(uint32_t* lock, uint32_t val) {
    asm volatile("red.relaxed.gpu.global.xor.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicAndKernel(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                     const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_and<>());
// CHECK-NEXT: }
__global__ void atomicAndKernel(uint32_t* lock, uint32_t val) {
    asm volatile("red.relaxed.gpu.global.and.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicMaxKernel(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::maximum<>());
// CHECK-NEXT: }
__global__ void atomicMaxKernel(uint32_t* lock, uint32_t val) {
    asm volatile("red.relaxed.gpu.global.max.u32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicMinKernel(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::minimum<>());
// CHECK-NEXT: }
__global__ void atomicMinKernel(uint32_t* lock, uint32_t val) {
    asm volatile("red.relaxed.gpu.global.min.u32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicAddKernelRelease(int* lock, int val,
// CHECK-NEXT:                        const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::plus<>());
// CHECK-NEXT:}
__global__ void atomicAddKernelRelease(int* lock, int val) {
    asm volatile("red.release.gpu.global.add.s32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicOrKernelRelease(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                     const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_or<>());
// CHECK-NEXT:}
__global__ void atomicOrKernelRelease(uint32_t* lock, uint32_t val) {
    asm volatile("red.release.gpu.global.or.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicXorKernelRelease(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_xor<>());
// CHECK-NEXT:}
__global__ void atomicXorKernelRelease(uint32_t* lock, uint32_t val) {
    asm volatile("red.release.gpu.global.xor.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicAndKernelRelease(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                     const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::bit_and<>());
// CHECK-NEXT: }
__global__ void atomicAndKernelRelease(uint32_t* lock, uint32_t val) {
    asm volatile("red.release.gpu.global.and.b32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicMaxKernelRelease(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::maximum<>());
// CHECK-NEXT: }
__global__ void atomicMaxKernelRelease(uint32_t* lock, uint32_t val) {
    asm volatile("red.release.gpu.global.max.u32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}

// CHECK: void atomicMinKernelRelease(uint32_t* lock, uint32_t val,
// CHECK-NEXT:                      const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:     *lock = sycl::reduce_over_group(item_ct1.get_group(), val,sycl::minimum<>());
// CHECK-NEXT: }
__global__ void atomicMinKernelRelease(uint32_t* lock, uint32_t val) {
    asm volatile("red.release.gpu.global.min.u32 [%0], %1;\n"
                 ::"l"(lock),"r"(val):"memory");
}
// clang-format on
